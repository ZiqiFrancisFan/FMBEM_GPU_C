#include "hip/hip_runtime.h"
/*
 * To change this license header, choose License Headers in Project Properties.
 * To change this template file, choose Tools | Templates
 * and open the template in the editor.
 */
#include "integral.h"
#include "translation.h"
#include "structs.h"

__constant__ float density = 1.2041;

__constant__ float speed = 343.21;

//Integral points and weights
__constant__ float INTPT[INTORDER]; 

__constant__ float INTWGT[INTORDER];

int genGaussParams(const int n, float *pt, float *wgt)
{
    int i, j;
    double t;
    gsl_vector *v = gsl_vector_alloc(n);
    for(i=0;i<n-1;i++) {
        gsl_vector_set(v,i,sqrt(pow(2*(i+1),2)-1));
    }
    for(i=0;i<n-1;i++) {
        t = gsl_vector_get(v,i);
        gsl_vector_set(v,i,(i+1)/t);
    }
    gsl_matrix *A = gsl_matrix_alloc(n,n);
    gsl_matrix *B = gsl_matrix_alloc(n,n);
    for(i=0;i<n;i++) {
        for(j=0;j<n;j++) {
            gsl_matrix_set(A,i,j,0);
            if(i==j) {
                gsl_matrix_set(B,i,j,1);
            } else {
                gsl_matrix_set(B,i,j,0);
            }
        }
    }
    for(i=0;i<n-1;i++) {
        t = gsl_vector_get(v,i);
        gsl_matrix_set(A,i+1,i,t);
        gsl_matrix_set(A,i,i+1,t);
    }
    gsl_eigen_symmv_workspace * wsp = gsl_eigen_symmv_alloc(n);
    HOST_CALL(gsl_eigen_symmv(A,v,B,wsp));
    for(i=0;i<n;i++) {
        pt[i] = gsl_vector_get(v,i);
        t = gsl_matrix_get(B,0,i);
        wgt[i] = 2*pow(t,2);
    }
    gsl_vector_free(v);
    gsl_matrix_free(A);
    gsl_matrix_free(B);
    return EXIT_SUCCESS;
}

int gaussPtsToDevice(const float *pt, const float *wgt)
{
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(INTPT),pt,INTORDER*sizeof(float),0,hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(INTWGT),wgt,INTORDER*sizeof(float),0,hipMemcpyHostToDevice));
    return EXIT_SUCCESS;
}

__host__ __device__ hipFloatComplex green(const float k, const cartCoord x, const cartCoord y)
{
    float r = cartDist(x,y);
    hipFloatComplex numerator = cplxExp(k*r);
    float denomenator = 4*PI*r;
    return make_hipFloatComplex(hipCrealf(numerator)/denomenator,hipCimagf(numerator)/denomenator);
}

__host__ __device__ hipFloatComplex triElemIntegral_G_nsgl(const float wavNum, const cartCoord nod[3], 
        const cartCoord y, const float *pt, const float *wgt)
{
    float J = cartNorm(crossProd(cartCoordSub(nod[0],nod[2]),cartCoordSub(nod[1],nod[2])));
    float rho, theta, eta1, eta2, xi1, xi2, xi3, wn, wm, temp;
    hipFloatComplex g, result = make_hipFloatComplex(0,0);
    cartCoord x;
    for(int n=0;n<INTORDER;n++) {
        eta2 = pt[n];
        wn = wgt[n];
        theta = 0.5f+0.5f*eta2;
        for(int m=0;m<INTORDER;m++) {
            eta1 = pt[m];
            wm = wgt[m];
            rho = 0.5f+0.5f*eta1;
            temp = 0.25f*wn*wm*rho*J;
            xi1 = rho*(1-theta);
            xi2 = rho-xi1;
            xi3 = 1-xi1-xi2;
            x = {
                    nod[0].x*xi1+nod[1].x*xi2+nod[2].x*xi3,
                    nod[0].y*xi1+nod[1].y*xi2+nod[2].y*xi3,
                    nod[0].z*xi1+nod[1].z*xi2+nod[2].z*xi3
                };
            g = green(wavNum,x,y);
            result = hipCaddf(result,make_hipFloatComplex(temp*hipCrealf(g),temp*hipCimagf(g)));
        }
    }
    return result;
}

__device__ hipFloatComplex triElemIntegral_G_nsgl(const float wavNum, const cartCoord nod[3], const cartCoord y)
{
    float J = cartNorm(crossProd(cartCoordSub(nod[0],nod[2]),cartCoordSub(nod[1],nod[2])));
    float rho, theta, eta1, eta2, xi1, xi2, xi3, wn, wm, temp;
    hipFloatComplex g, result = make_hipFloatComplex(0,0);
    cartCoord x;
    for(int n=0;n<INTORDER;n++) {
        eta2 = INTPT[n];
        wn = INTWGT[n];
        theta = 0.5f+0.5f*eta2;
        for(int m=0;m<INTORDER;m++) {
            eta1 = INTPT[m];
            wm = INTWGT[m];
            rho = 0.5f+0.5f*eta1;
            temp = 0.25f*wn*wm*rho*J;
            xi1 = rho*(1-theta);
            xi2 = rho-xi1;
            xi3 = 1-xi1-xi2;
            x = {
                    nod[0].x*xi1+nod[1].x*xi2+nod[2].x*xi3,
                    nod[0].y*xi1+nod[1].y*xi2+nod[2].y*xi3,
                    nod[0].z*xi1+nod[1].z*xi2+nod[2].z*xi3
                };
            g = green(wavNum,x,y);
            result = hipCaddf(result,make_hipFloatComplex(temp*hipCrealf(g),temp*hipCimagf(g)));
        }
    }
    return result;
}

__host__ __device__ hipFloatComplex triElemIntegral_G_sgl_3(const float wavNum, const cartCoord nod[3], 
        const float *pt, const float *wgt)
{
    float J = cartNorm(crossProd(cartCoordSub(nod[0],nod[2]),cartCoordSub(nod[1],nod[2])));
    float rho, theta, eta1, eta2, xi1, xi2, xi3, wn, wm, temp;
    hipFloatComplex g, result = make_hipFloatComplex(0,0);
    cartCoord x;
    for(int n=0;n<INTORDER;n++) {
        eta2 = pt[n];
        wn = wgt[n];
        theta = 0.5f+0.5f*eta2;
        for(int m=0;m<INTORDER;m++) {
            eta1 = pt[m];
            wm = wgt[m];
            rho = 0.5f+0.5f*eta1;
            temp = 0.25f*wn*wm*rho*J;
            xi1 = rho*(1-theta);
            xi2 = rho-xi1;
            xi3 = 1-xi1-xi2;
            x = {
                    nod[0].x*xi1+nod[1].x*xi2+nod[2].x*xi3,
                    nod[0].y*xi1+nod[1].y*xi2+nod[2].y*xi3,
                    nod[0].z*xi1+nod[1].z*xi2+nod[2].z*xi3
                };
            g = green(wavNum,x,nod[2]);
            result = hipCaddf(result,make_hipFloatComplex(temp*hipCrealf(g),temp*hipCimagf(g)));
        }
    }
    return result;
}

__device__ hipFloatComplex triElemIntegral_G_sgl_3(const float wavNum, const cartCoord nod[3])
{
    float J = cartNorm(crossProd(cartCoordSub(nod[0],nod[2]),cartCoordSub(nod[1],nod[2])));
    float rho, theta, eta1, eta2, xi1, xi2, xi3, wn, wm, temp;
    hipFloatComplex g, result = make_hipFloatComplex(0,0);
    cartCoord x;
    for(int n=0;n<INTORDER;n++) {
        eta2 = INTPT[n];
        wn = INTWGT[n];
        theta = 0.5f+0.5f*eta2;
        for(int m=0;m<INTORDER;m++) {
            eta1 = INTPT[m];
            wm = INTWGT[m];
            rho = 0.5f+0.5f*eta1;
            temp = 0.25f*wn*wm*rho*J;
            xi1 = rho*(1-theta);
            xi2 = rho-xi1;
            xi3 = 1-xi1-xi2;
            x = {
                    nod[0].x*xi1+nod[1].x*xi2+nod[2].x*xi3,
                    nod[0].y*xi1+nod[1].y*xi2+nod[2].y*xi3,
                    nod[0].z*xi1+nod[1].z*xi2+nod[2].z*xi3
                };
            g = green(wavNum,x,nod[2]);
            result = hipCaddf(result,make_hipFloatComplex(temp*hipCrealf(g),temp*hipCimagf(g)));
        }
    }
    return result;
}

__host__ __device__ hipFloatComplex triElemIntegral_G_sgl(const float wavNum, const cartCoord nod[3], 
        const float *pt, const float *wgt)
{
    cartCoord y;
    cartCoord ctr23 = scalarMul(0.5,cartCoordAdd(nod[1],nod[2]));
    y = cartCoordAdd(nod[0],scalarMul(2.0f/3.0f,cartCoordSub(ctr23,nod[0])));
    hipFloatComplex result1 = make_hipFloatComplex(0,0), result2 = make_hipFloatComplex(0,0), 
            result3 = make_hipFloatComplex(0,0), result = make_hipFloatComplex(0,0);
    cartCoord nod_sub[3];
    nod_sub[0] = nod[0];
    nod_sub[1] = nod[1];
    nod_sub[2] = y;
    result1 = triElemIntegral_G_sgl_3(wavNum,nod_sub,pt,wgt);
    nod_sub[0] = nod[1];
    nod_sub[1] = nod[2];
    nod_sub[2] = y;
    result2 = triElemIntegral_G_sgl_3(wavNum,nod_sub,pt,wgt);
    nod_sub[0] = nod[2];
    nod_sub[1] = nod[0];
    nod_sub[2] = y;
    result3 = triElemIntegral_G_sgl_3(wavNum,nod_sub,pt,wgt);
    result = hipCaddf(hipCaddf(result1,result2),result3);
    return result;
}

__device__ hipFloatComplex triElemIntegral_G_sgl(const float wavNum, const cartCoord nod[3])
{
    cartCoord y;
    cartCoord ctr23 = scalarMul(0.5,cartCoordAdd(nod[1],nod[2]));
    y = cartCoordAdd(nod[0],scalarMul(2.0f/3.0f,cartCoordSub(ctr23,nod[0])));
    hipFloatComplex result1 = make_hipFloatComplex(0,0), result2 = make_hipFloatComplex(0,0), 
            result3 = make_hipFloatComplex(0,0), result = make_hipFloatComplex(0,0);
    cartCoord nod_sub[3];
    nod_sub[0] = nod[0];
    nod_sub[1] = nod[1];
    nod_sub[2] = y;
    result1 = triElemIntegral_G_sgl_3(wavNum,nod_sub);
    nod_sub[0] = nod[1];
    nod_sub[1] = nod[2];
    nod_sub[2] = y;
    result2 = triElemIntegral_G_sgl_3(wavNum,nod_sub);
    nod_sub[0] = nod[2];
    nod_sub[1] = nod[0];
    nod_sub[2] = y;
    result3 = triElemIntegral_G_sgl_3(wavNum,nod_sub);
    result = hipCaddf(hipCaddf(result1,result2),result3);
    return result;
}

__host__ hipFloatComplex triElemIntegral_R(const float wavNum, const cartCoord nod[3], 
        const int n, const int m, const cartCoord x_lp, const float *pt, const float *wgt)
{
    float J = cartNorm(crossProd(cartCoordSub(nod[0],nod[2]),cartCoordSub(nod[1],nod[2])));
    float rho, theta, eta1, eta2, xi1, xi2, xi3, wn, wm, temp;
    hipFloatComplex rgl, result = make_hipFloatComplex(0,0);
    cartCoord x;
    sphCoord sphTemp;
    for(int i=0;i<INTORDER;i++) {
        eta2 = pt[i];
        wn = wgt[i];
        theta = 0.5f+0.5f*eta2;
        for(int j=0;j<INTORDER;j++) {
            eta1 = pt[j];
            wm = wgt[j];
            rho = 0.5f+0.5f*eta1;
            temp = 0.25f*wn*wm*rho*J;
            xi1 = rho*(1-theta);
            xi2 = rho-xi1;
            xi3 = 1-xi1-xi2;
            x = {
                    nod[0].x*xi1+nod[1].x*xi2+nod[2].x*xi3,
                    nod[0].y*xi1+nod[1].y*xi2+nod[2].y*xi3,
                    nod[0].z*xi1+nod[1].z*xi2+nod[2].z*xi3
                };
            sphTemp = cart2sph(cartCoordSub(x,x_lp));
            rgl = gsl_complex2cuFloatComplex(rglBasis(wavNum,n,m,sphTemp));
            result = hipCaddf(result,make_hipFloatComplex(temp*hipCrealf(rgl),temp*hipCimagf(rgl)));
        }
    }
    return result;
}

__host__ hipFloatComplex dirDev_R(const float wavNum, const int n, const int m, 
        const cartCoord nrml, const cartCoord x) 
{
    hipFloatComplex temp_c[6];
    float temp_f[6];
    sphCoord coord_sph = cart2sph(x);
    temp_c[0] = gsl_complex2cuFloatComplex(rglBasis(wavNum,n+1,m+1,coord_sph));
    if(n-1<0) {
        temp_c[1] = make_hipFloatComplex(0,0);
    } else {
        temp_c[1] = gsl_complex2cuFloatComplex(rglBasis(wavNum,n-1,m+1,coord_sph));
    }
    temp_c[2] = gsl_complex2cuFloatComplex(rglBasis(wavNum,n+1,m-1,coord_sph));
    if(n-1<0) {
        temp_c[3] = make_hipFloatComplex(0,0);
    } else {
        temp_c[3] = gsl_complex2cuFloatComplex(rglBasis(wavNum,n-1,m-1,coord_sph));
    }
    if(n-1<0) {
        temp_c[4] = make_hipFloatComplex(0,0);
    } else {
        temp_c[4] = gsl_complex2cuFloatComplex(rglBasis(wavNum,n-1,m,coord_sph));
    } 
    temp_c[5] = gsl_complex2cuFloatComplex(rglBasis(wavNum,n+1,m,coord_sph));
    temp_f[0] = bCoeff(n+1,-(m+1));
    temp_f[1] = bCoeff(n,m);
    temp_f[2] = bCoeff(n+1,m-1);
    temp_f[3] = bCoeff(n,-m);
    temp_f[4] = aCoeff(n-1,m);
    temp_f[5] = aCoeff(n,m);
    temp_c[0] = make_hipFloatComplex(hipCrealf(temp_c[0])*temp_f[0],hipCimagf(temp_c[0])*temp_f[0]);
    temp_c[1] = make_hipFloatComplex(hipCrealf(temp_c[1])*temp_f[1],hipCimagf(temp_c[1])*temp_f[1]);
    temp_c[2] = make_hipFloatComplex(hipCrealf(temp_c[2])*temp_f[2],hipCimagf(temp_c[2])*temp_f[2]);
    temp_c[3] = make_hipFloatComplex(hipCrealf(temp_c[3])*temp_f[3],hipCimagf(temp_c[3])*temp_f[3]);
    temp_c[4] = make_hipFloatComplex(hipCrealf(temp_c[4])*temp_f[4],hipCimagf(temp_c[4])*temp_f[4]);
    temp_c[5] = make_hipFloatComplex(hipCrealf(temp_c[5])*temp_f[5],hipCimagf(temp_c[5])*temp_f[5]);
    temp_c[0] = hipCsubf(temp_c[0],temp_c[1]);
    temp_c[1] = hipCsubf(temp_c[2],temp_c[3]);
    temp_c[2] = hipCsubf(temp_c[4],temp_c[5]);
    temp_c[3] = make_hipFloatComplex(wavNum/2.0f*nrml.x,-wavNum/2.0f*nrml.y);
    temp_c[4] = make_hipFloatComplex(wavNum/2.0f*nrml.x,wavNum/2.0f*nrml.y);
    temp_c[0] = hipCmulf(temp_c[3],temp_c[0]);
    temp_c[1] = hipCmulf(temp_c[4],temp_c[1]);
    temp_c[2] = make_hipFloatComplex(wavNum*nrml.z*hipCrealf(temp_c[2]),wavNum*nrml.z*hipCimagf(temp_c[2]));
    temp_c[0] = hipCaddf(temp_c[0],temp_c[1]);
    temp_c[0] = hipCaddf(temp_c[0],temp_c[2]);
    return temp_c[0];
}

__host__ hipFloatComplex triElemIntegral_pRpn(const float wavNum, const cartCoord nod[3], 
        const int n, const int m, const cartCoord x_lp, const float *pt, const float *wgt)
{
    cartCoord nrml = crossProd(cartCoordSub(nod[0],nod[2]),cartCoordSub(nod[1],nod[2]));
    cartCoord nrml_nrmlzd = normalize(nrml);
    float J = cartNorm(nrml);
    float rho, theta, eta1, eta2, xi1, xi2, xi3, wn, wm, temp;
    hipFloatComplex pRpn, result = make_hipFloatComplex(0,0);
    cartCoord x;
    for(int i=0;i<INTORDER;i++) {
        eta2 = pt[i];
        wn = wgt[i];
        theta = 0.5f+0.5f*eta2;
        for(int j=0;j<INTORDER;j++) {
            eta1 = pt[j];
            wm = wgt[j];
            rho = 0.5f+0.5f*eta1;
            temp = 0.25f*wn*wm*rho*J;
            xi1 = rho*(1-theta);
            xi2 = rho-xi1;
            xi3 = 1-xi1-xi2;
            x = {
                    nod[0].x*xi1+nod[1].x*xi2+nod[2].x*xi3,
                    nod[0].y*xi1+nod[1].y*xi2+nod[2].y*xi3,
                    nod[0].z*xi1+nod[1].z*xi2+nod[2].z*xi3
                };
            x = cartCoordSub(x,x_lp);
            pRpn = dirDev_R(wavNum,n,m,nrml_nrmlzd,x);
            result = hipCaddf(result,make_hipFloatComplex(temp*hipCrealf(pRpn),temp*hipCimagf(pRpn)));
        }
    }
    return result;
}

__host__ __device__ float prpn1(const cartCoord n, const cartCoord x, const cartCoord y)
{
    float dist = cartDist(x,y);
    return ((x.x-y.x)*n.x+(x.y-y.y)*n.y+(x.z-y.z)*n.z)/dist;
}

__host__ __device__ float prpn2(const cartCoord n, const cartCoord x, const cartCoord y)
{
    float dist = cartDist(x,y);
    return ((y.x-x.x)*n.x+(y.y-x.y)*n.y+(y.z-x.z)*n.z)/dist;
}

__host__ __device__ float prRecippn1(const cartCoord n, const cartCoord x, const cartCoord y)
{
    float dist = cartDist(x,y);
    return -1.0f/(dist*dist)*prpn1(n,x,y);
}

__host__ __device__ float prRecippn2(const cartCoord n, const cartCoord x, const cartCoord y)
{
    float dist = cartDist(x,y);
    return -1.0f/(dist*dist)*prpn2(n,x,y);
}

__host__ __device__ hipFloatComplex pGp1n(const float wavNum, const cartCoord x, const cartCoord y, 
        const cartCoord n)
{
    hipFloatComplex temp_c[2];
    float temp_f, dist = cartDist(x,y);
    temp_c[0] = green(wavNum,x,y);
    temp_c[1] = make_hipFloatComplex(-1.0f/dist,wavNum);
    temp_f = prpn1(n,x,y);
    temp_c[0] = hipCmulf(temp_c[0],temp_c[1]);
    temp_c[0] = make_hipFloatComplex(hipCrealf(temp_c[0])*temp_f,hipCimagf(temp_c[0])*temp_f);
    return temp_c[0];
}

__host__ __device__ hipFloatComplex pGp2n(const float wavNum, const cartCoord n, 
        const cartCoord x, const cartCoord y)
{
    hipFloatComplex temp_c[2];
    float temp_f, dist = cartDist(x,y);
    temp_c[0] = green(wavNum,x,y);
    temp_c[1] = make_hipFloatComplex(-1.0f/dist,wavNum);
    temp_f = prpn2(n,x,y);
    temp_c[0] = hipCmulf(temp_c[0],temp_c[1]);
    temp_c[0] = make_hipFloatComplex(hipCrealf(temp_c[0])*temp_f,hipCimagf(temp_c[0])*temp_f);
    return temp_c[0];
}

__host__ __device__ hipFloatComplex triElemIntegral_pGp1n_nsgl(const float wavNum, 
        const cartCoord nod[3], const cartCoord y, const float *pt, const float *wgt)
{
    cartCoord nrml = crossProd(cartCoordSub(nod[0],nod[2]),cartCoordSub(nod[1],nod[2]));
    cartCoord nrml_nrmlzd = normalize(nrml);
    float J = cartNorm(nrml);
    float rho, theta, eta1, eta2, xi1, xi2, xi3, wn, wm, temp;
    hipFloatComplex pgpn1, result = make_hipFloatComplex(0,0);
    cartCoord x;
    for(int n=0;n<INTORDER;n++) {
        eta2 = pt[n];
        wn = wgt[n];
        theta = 0.5f+0.5f*eta2;
        for(int m=0;m<INTORDER;m++) {
            eta1 = pt[m];
            wm = wgt[m];
            rho = 0.5f+0.5f*eta1;
            temp = 0.25f*wn*wm*rho*J;
            xi1 = rho*(1-theta);
            xi2 = rho-xi1;
            xi3 = 1-xi1-xi2;
            x = {
                    nod[0].x*xi1+nod[1].x*xi2+nod[2].x*xi3,
                    nod[0].y*xi1+nod[1].y*xi2+nod[2].y*xi3,
                    nod[0].z*xi1+nod[1].z*xi2+nod[2].z*xi3
                };
            pgpn1 = pGp1n(wavNum,x,y,nrml_nrmlzd);
            result = hipCaddf(result,make_hipFloatComplex(temp*hipCrealf(pgpn1),
                    temp*hipCimagf(pgpn1)));
        }
    }
    return result;
}

__host__ __device__ hipFloatComplex triElemIntegral_pGp1n_sgl_3(const float wavNum, const cartCoord nod[3], 
        const float *pt, const float *wgt)
{
    cartCoord nrml = crossProd(cartCoordSub(nod[0],nod[2]),cartCoordSub(nod[1],nod[2]));
    cartCoord nrml_nrmlzd = normalize(nrml);
    float J = cartNorm(nrml);
    float rho, theta, eta1, eta2, xi1, xi2, xi3, wn, wm, temp;
    hipFloatComplex pgpn1, result = make_hipFloatComplex(0,0);
    cartCoord x;
    for(int n=0;n<INTORDER;n++) {
        eta2 = pt[n];
        wn = wgt[n];
        theta = 0.5f+0.5f*eta2;
        for(int m=0;m<INTORDER;m++) {
            eta1 = pt[m];
            wm = wgt[m];
            rho = 0.5f+0.5f*eta1;
            temp = 0.25f*wn*wm*rho*J;
            xi1 = rho*(1-theta);
            xi2 = rho-xi1;
            xi3 = 1-xi1-xi2;
            x = {
                    nod[0].x*xi1+nod[1].x*xi2+nod[2].x*xi3,
                    nod[0].y*xi1+nod[1].y*xi2+nod[2].y*xi3,
                    nod[0].z*xi1+nod[1].z*xi2+nod[2].z*xi3
                };
            pgpn1 = pGp1n(wavNum,x,nod[2],nrml_nrmlzd);
            result = hipCaddf(result,make_hipFloatComplex(temp*hipCrealf(pgpn1),
                    temp*hipCimagf(pgpn1)));
        }
    }
    return result;
}

__host__ __device__ hipFloatComplex triElemIntegral_pGp1n_sgl(const float wavNum, const cartCoord nod[3], 
        const float *pt, const float *wgt)
{
    cartCoord y;
    cartCoord ctr23 = scalarMul(0.5,cartCoordAdd(nod[1],nod[2]));
    y = cartCoordAdd(nod[0],scalarMul(2.0f/3.0f,cartCoordSub(ctr23,nod[0])));
    hipFloatComplex result1 = make_hipFloatComplex(0,0), result2 = make_hipFloatComplex(0,0), 
            result3 = make_hipFloatComplex(0,0), result = make_hipFloatComplex(0,0);
    cartCoord nod_sub[3];
    nod_sub[0] = nod[0];
    nod_sub[1] = nod[1];
    nod_sub[2] = y;
    result1 = triElemIntegral_pGp1n_sgl_3(wavNum,nod_sub,pt,wgt);
    nod_sub[0] = nod[1];
    nod_sub[1] = nod[2];
    nod_sub[2] = y;
    result2 = triElemIntegral_pGp1n_sgl_3(wavNum,nod_sub,pt,wgt);
    nod_sub[0] = nod[2];
    nod_sub[1] = nod[0];
    nod_sub[2] = y;
    result3 = triElemIntegral_pGp1n_sgl_3(wavNum,nod_sub,pt,wgt);
    result = hipCaddf(hipCaddf(result1,result2),result3);
    return result;
}

__host__ __device__ hipFloatComplex triElemIntegral_pGp2n_nsgl(const float wavNum, 
        const cartCoord nod[3], const cartCoord nrml, const cartCoord y, 
        const float *pt, const float *wgt)
{
    float J = cartNorm(crossProd(cartCoordSub(nod[0],nod[2]),cartCoordSub(nod[1],nod[2])));
    float rho, theta, eta1, eta2, xi1, xi2, xi3, wn, wm, temp;
    hipFloatComplex pgpn2, result = make_hipFloatComplex(0,0);
    cartCoord x;
    for(int n=0;n<INTORDER;n++) {
        eta2 = pt[n];
        wn = wgt[n];
        theta = 0.5f+0.5f*eta2;
        for(int m=0;m<INTORDER;m++) {
            eta1 = pt[m];
            wm = wgt[m];
            rho = 0.5f+0.5f*eta1;
            temp = 0.25f*wn*wm*rho*J;
            xi1 = rho*(1-theta);
            xi2 = rho-xi1;
            xi3 = 1-xi1-xi2;
            x = {
                    nod[0].x*xi1+nod[1].x*xi2+nod[2].x*xi3,
                    nod[0].y*xi1+nod[1].y*xi2+nod[2].y*xi3,
                    nod[0].z*xi1+nod[1].z*xi2+nod[2].z*xi3
                };
            pgpn2 = pGp2n(wavNum,nrml,x,y);
            result = hipCaddf(result,make_hipFloatComplex(temp*hipCrealf(pgpn2),
                    temp*hipCimagf(pgpn2)));
        }
    }
    return result;
}

__host__ __device__ hipFloatComplex triElemIntegral_pGp2n_sgl_3(const float wavNum, 
        const cartCoord nod[3], const float *pt, const float *wgt)
{
    cartCoord nrml = crossProd(cartCoordSub(nod[0],nod[2]),cartCoordSub(nod[1],nod[2]));
    cartCoord nrml_nrmlzd = normalize(nrml);
    float J = cartNorm(nrml);
    float rho, theta, eta1, eta2, xi1, xi2, xi3, wn, wm, temp;
    hipFloatComplex pgpn2, result = make_hipFloatComplex(0,0);
    cartCoord x;
    for(int n=0;n<INTORDER;n++) {
        eta2 = pt[n];
        wn = wgt[n];
        theta = 0.5f+0.5f*eta2;
        for(int m=0;m<INTORDER;m++) {
            eta1 = pt[m];
            wm = wgt[m];
            rho = 0.5f+0.5f*eta1;
            temp = 0.25f*wn*wm*rho*J;
            xi1 = rho*(1-theta);
            xi2 = rho-xi1;
            xi3 = 1-xi1-xi2;
            x = {
                    nod[0].x*xi1+nod[1].x*xi2+nod[2].x*xi3,
                    nod[0].y*xi1+nod[1].y*xi2+nod[2].y*xi3,
                    nod[0].z*xi1+nod[1].z*xi2+nod[2].z*xi3
                };
            pgpn2 = pGp2n(wavNum,nrml_nrmlzd,x,nod[2]);
            result = hipCaddf(result,make_hipFloatComplex(temp*hipCrealf(pgpn2),
                    temp*hipCimagf(pgpn2)));
        }
    }
    return result;
}

__host__ __device__ hipFloatComplex triElemIntegral_pGp2n_sgl(const float wavNum, 
        const cartCoord nod[3], const float *pt, const float *wgt)
{
    cartCoord y;
    cartCoord ctr23 = scalarMul(0.5,cartCoordAdd(nod[1],nod[2]));
    y = cartCoordAdd(nod[0],scalarMul(2.0f/3.0f,cartCoordSub(ctr23,nod[0])));
    hipFloatComplex result1 = make_hipFloatComplex(0,0), result2 = make_hipFloatComplex(0,0), 
            result3 = make_hipFloatComplex(0,0), result = make_hipFloatComplex(0,0);
    cartCoord nod_sub[3];
    nod_sub[0] = nod[0];
    nod_sub[1] = nod[1];
    nod_sub[2] = y;
    result1 = triElemIntegral_pGp2n_sgl_3(wavNum,nod_sub,pt,wgt);
    nod_sub[0] = nod[1];
    nod_sub[1] = nod[2];
    nod_sub[2] = y;
    result2 = triElemIntegral_pGp2n_sgl_3(wavNum,nod_sub,pt,wgt);
    nod_sub[0] = nod[2];
    nod_sub[1] = nod[0];
    nod_sub[2] = y;
    result3 = triElemIntegral_pGp2n_sgl_3(wavNum,nod_sub,pt,wgt);
    result = hipCaddf(hipCaddf(result1,result2),result3);
    return result;
}

__host__ __device__ hipFloatComplex p2Gp1np2n(const float wavNum, const cartCoord n1, const cartCoord n2, 
        const cartCoord x, const cartCoord y)
{
    hipFloatComplex temp[3];
    float dist, t;
    
    dist = cartDist(x,y);
    temp[0] = cplxExp(wavNum*dist);
    t = 4*PI*dist*dist*dist;
    temp[0] = make_hipFloatComplex(hipCrealf(temp[0])/t,hipCimagf(temp[0])/t);
    
    temp[1] = make_hipFloatComplex(3-wavNum*wavNum*dist*dist,-3*wavNum*dist);
    t = prpn1(n1,x,y)*prpn2(n2,x,y);
    temp[1] = make_hipFloatComplex(hipCrealf(temp[1])*t,hipCimagf(temp[1])*t);
    
    t = dotProd(n1,n2);
    temp[2] = make_hipFloatComplex(1,-wavNum*dist);
    temp[2] = make_hipFloatComplex(hipCrealf(temp[2])*t,hipCimagf(temp[2])*t);
    
    return hipCmulf(temp[0],hipCaddf(temp[1],temp[2]));
}

__host__ __device__ hipFloatComplex triElemIntegral_p2Gp1np2n_nsgl(const float wavNum, 
        const cartCoord nod[3], const cartCoord nrml_y, const cartCoord y, 
        const float *pt, const float *wgt)
{
    cartCoord nrml_x = crossProd(cartCoordSub(nod[0],nod[2]),cartCoordSub(nod[1],nod[2]));
    cartCoord nrml_x_nrmlzd = normalize(nrml_x);
    float J = cartNorm(nrml_x);
    float rho, theta, eta1, eta2, xi1, xi2, xi3, wn, wm, temp;
    hipFloatComplex p2gpn1pn2, result = make_hipFloatComplex(0,0);
    cartCoord x;
    for(int n=0;n<INTORDER;n++) {
        eta2 = pt[n];
        wn = wgt[n];
        theta = 0.5f+0.5f*eta2;
        for(int m=0;m<INTORDER;m++) {
            eta1 = pt[m];
            wm = wgt[m];
            rho = 0.5f+0.5f*eta1;
            temp = 0.25f*wn*wm*rho*J;
            xi1 = rho*(1-theta);
            xi2 = rho-xi1;
            xi3 = 1-xi1-xi2;
            x = {
                    nod[0].x*xi1+nod[1].x*xi2+nod[2].x*xi3,
                    nod[0].y*xi1+nod[1].y*xi2+nod[2].y*xi3,
                    nod[0].z*xi1+nod[1].z*xi2+nod[2].z*xi3
                };
            p2gpn1pn2 = p2Gp1np2n(wavNum,nrml_x_nrmlzd,nrml_y,x,y);
            result = hipCaddf(result,make_hipFloatComplex(temp*hipCrealf(p2gpn1pn2),
                    temp*hipCimagf(p2gpn1pn2)));
        }
    }
    return result;
}

__host__ __device__ cartCoord triCentroid(const cartCoord nod[3])
{
    cartCoord ctr_23 = scalarMul(0.5,cartCoordAdd(nod[1],nod[2]));
    cartCoord centroid = cartCoordAdd(nod[0],scalarMul(2.0/3.0,cartCoordSub(ctr_23,nod[0])));
    return centroid;
}

__host__ __device__ float theta2rho(const cartCoord nod[3], const float theta) 
{
    cartCoord vc = triCentroid(nod);
    float t, rho;
    //c1 is the polar axis
    float d_c1 = cartDist(vc,nod[0]); //distance between center and node 1
    float d_c2 = cartDist(vc,nod[1]); //distance between center and node 2
    float d_c3 = cartDist(vc,nod[2]); //distance between center and node 3
    float d_12 = cartDist(nod[0],nod[1]); //distance between node 1 and node 2
    float d_23 = cartDist(nod[1],nod[2]); //distance between node 2 and node 3
    float d_31 = cartDist(nod[2],nod[0]); //distance between node 3 and node 1
    //float theta_1 = 0;
    t = (powf(d_c1,2)+powf(d_c2,2)-powf(d_12,2))/(2*d_c1*d_c2);
    float theta_2 = acosf(t); //angle 1c2
    t = (powf(d_c2,2)+powf(d_c3,2)-powf(d_23,2))/(2*d_c2*d_c3);
    float theta_3 = theta_2+acosf(t); //angle 1c3 in the order of 1c2 and 2c3
    //coordinate of node 1 in the local coordinate system
    float x_1 = d_c1;
    float y_1 = 0;
    //coordinate of node 2 in the local coordinate system
    float x_2 = d_c2*cosf(theta_2);
    float y_2 = d_c2*sinf(theta_2);
    //coordinate of node 3 in the local coordinate system
    float x_3 = d_c3*cosf(theta_3);
    float y_3 = d_c3*sinf(theta_3);
    float x_i, y_i; //intersection point
    float k; //slope
    
    if(theta<theta_2) {
        //the intersection point is with line 12
        if(theta==0.5*PI) {
            x_i = 0;
            y_i = (y_2-y_1)/(x_2-x_1)*(-x_1)+y_1;
        } else {
            k = tanf(theta);
            x_i = (y_1-(y_2-y_1)/(x_2-x_1)*x_1)/(k-(y_2-y_1)/(x_2-x_1));
            y_i = k*x_i;
        }
    }
    if(theta>=theta_2 && theta<theta_3) {
        //intersects with line 32
        if(theta==0.5*PI) {
            x_i = 0;
            y_i = (y_3-y_2)/(x_3-x_2)*(-x_2)+y_2;
        } else {
            k = tan(theta);
            x_i = (y_2-(y_3-y_2)/(x_3-x_2)*x_2)/(k-(y_3-y_2)/(x_3-x_2));
            y_i = k*x_i;
        }
    }
    if(theta>=theta_3) {
        //intersects with line 21
        if(theta==0.5*PI) {
            x_i = 0;
            y_i = (y_1-y_3)/(x_1-x_3)*(-x_3)+y_3;
        } else {
            k = tan(theta);
            x_i = (y_3-(y_1-y_3)/(x_1-x_3)*x_3)/(k-(y_1-y_3)/(x_1-x_3));
            y_i = k*x_i;
        }
    }
    rho = sqrtf(powf(x_i,2)+powf(y_i,2));
    return rho;
}

__host__ __device__ hipFloatComplex triElemIntegral_p2Gp1np2n_sgl(const float wavNum, 
        const cartCoord nod[3], const float *pt, const float *wgt)
{
    float theta, rho, s, w;
    hipFloatComplex sum, t;
    sum = make_hipFloatComplex(0,0);
    for(int i=0;i<INTORDER;i++) {
        s = pt[i];
        w = wgt[i];
        theta = PI*s+PI;
        rho = theta2rho(nod,theta);
        t = make_hipFloatComplex(cosf(wavNum*rho),sinf(wavNum*rho));
        t = make_hipFloatComplex(1.0f/rho*hipCrealf(t),1.0f/rho*hipCimagf(t));
        t = make_hipFloatComplex(w*hipCrealf(t),w*hipCimagf(t));
        sum = hipCaddf(sum,t);
    }
    sum = make_hipFloatComplex(PI*hipCrealf(sum),PI*hipCimagf(sum));
    sum = hipCsubf(sum,make_hipFloatComplex(0,2*PI*wavNum));
    sum = make_hipFloatComplex(-1.0f/(4*PI)*hipCrealf(sum),-1.0f/(4*PI)*hipCimagf(sum));
    return sum;
}

__host__ __device__ void cmptDiffCoeff(const float wavNum, const hipFloatComplex *coeff, 
        const int p, const cartCoord nrml, hipFloatComplex *coeff_n)
{
    hipFloatComplex c[6], temp_c[6];
    float temp_f;
    for(int n=0;n<p;n++) {
        for(int m=-n;m<=n;m++) {
            if(abs(m-1)>n-1) {
                c[0] = make_hipFloatComplex(0,0);
            } else {
                c[0] = coeff[NM2IDX0(n-1,m-1)];
            }
            if(abs(m-1)>n+1) {
                c[1] = make_hipFloatComplex(0,0);
            } else {
                c[1] = coeff[NM2IDX0(n+1,m-1)];
            }
            if(abs(m+1)>n-1) {
                c[2] = make_hipFloatComplex(0,0);
            } else {
                c[2] = coeff[NM2IDX0(n-1,m+1)];
            }
            if(abs(m+1)>n+1) {
                c[3] = make_hipFloatComplex(0,0);
            } else {
                c[3] = coeff[NM2IDX0(n+1,m+1)];
            }
            if(abs(m)>n+1) {
                c[4] = make_hipFloatComplex(0,0);
            } else {
                c[4] = coeff[NM2IDX0(n+1,m)];
            }
            if(abs(m)>n-1) {
                c[5] = make_hipFloatComplex(0,0);
            } else {
                c[5] = coeff[NM2IDX0(n-1,m)];
            }
            temp_f = bCoeff(n,-m);
            temp_c[0] = make_hipFloatComplex(temp_f*hipCrealf(c[0]),temp_f*hipCimagf(c[0]));
            temp_f = bCoeff(n+1,m-1);
            temp_c[1] = make_hipFloatComplex(temp_f*hipCrealf(c[1]),temp_f*hipCimagf(c[1]));
            temp_f = bCoeff(n,m);
            temp_c[2] = make_hipFloatComplex(temp_f*hipCrealf(c[2]),temp_f*hipCimagf(c[2]));
            temp_f = bCoeff(n+1,-m-1);
            temp_c[2] = make_hipFloatComplex(temp_f*hipCrealf(c[3]),temp_f*hipCimagf(c[3]));
            temp_f = aCoeff(n,m);
            temp_c[2] = make_hipFloatComplex(temp_f*hipCrealf(c[4]),temp_f*hipCimagf(c[4]));
            temp_f = aCoeff(n-1,m);
            temp_c[2] = make_hipFloatComplex(temp_f*hipCrealf(c[5]),temp_f*hipCimagf(c[5]));
            
            temp_c[0] = hipCsubf(temp_c[0],temp_c[1]);
            temp_c[1] = hipCsubf(temp_c[2],temp_c[3]);
            temp_c[2] = hipCsubf(temp_c[4],temp_c[5]);
            
            temp_f = wavNum/2;
            temp_c[3] = make_hipFloatComplex(temp_f*nrml.x,-temp_f*nrml.y);
            temp_c[4] = make_hipFloatComplex(temp_f*nrml.x,temp_f*nrml.y);
            
            temp_c[0] = hipCmulf(temp_c[0],temp_c[3]);
            temp_c[1] = hipCmulf(temp_c[1],temp_c[4]);
            temp_f = wavNum*nrml.z;
            temp_c[2] = make_hipFloatComplex(temp_f*hipCrealf(temp_c[2]),temp_f*hipCimagf(temp_c[2]));
            
            coeff_n[NM2IDX0(n,m)] = hipCaddf(hipCaddf(temp_c[0],temp_c[1]),temp_c[2]);
        }
    }
}







