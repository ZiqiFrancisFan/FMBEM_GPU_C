#include "hip/hip_runtime.h"
/*
 * To change this license header, choose License Headers in Project Properties.
 * To change this template file, choose Tools | Templates
 * and open the template in the editor.
 */
#include "integral.h"
#include "translation.h"
#include "structs.h"

__constant__ float density = 1.2041;

__constant__ float speed = 343.21;

//Integral points and weights
__constant__ float INTPT[INTORDER]; 

__constant__ float INTWGT[INTORDER];

int genGaussParams(const int n, float *pt, float *wgt)
{
    int i, j;
    double t;
    gsl_vector *v = gsl_vector_alloc(n);
    for(i=0;i<n-1;i++) {
        gsl_vector_set(v,i,sqrt(pow(2*(i+1),2)-1));
    }
    for(i=0;i<n-1;i++) {
        t = gsl_vector_get(v,i);
        gsl_vector_set(v,i,(i+1)/t);
    }
    gsl_matrix *A = gsl_matrix_alloc(n,n);
    gsl_matrix *B = gsl_matrix_alloc(n,n);
    for(i=0;i<n;i++) {
        for(j=0;j<n;j++) {
            gsl_matrix_set(A,i,j,0);
            if(i==j) {
                gsl_matrix_set(B,i,j,1);
            } else {
                gsl_matrix_set(B,i,j,0);
            }
        }
    }
    for(i=0;i<n-1;i++) {
        t = gsl_vector_get(v,i);
        gsl_matrix_set(A,i+1,i,t);
        gsl_matrix_set(A,i,i+1,t);
    }
    gsl_eigen_symmv_workspace * wsp = gsl_eigen_symmv_alloc(n);
    HOST_CALL(gsl_eigen_symmv(A,v,B,wsp));
    for(i=0;i<n;i++) {
        pt[i] = gsl_vector_get(v,i);
        t = gsl_matrix_get(B,0,i);
        wgt[i] = 2*pow(t,2);
    }
    gsl_vector_free(v);
    gsl_matrix_free(A);
    gsl_matrix_free(B);
    return EXIT_SUCCESS;
}

int gaussPtsToDevice(const float *pt, const float *wgt)
{
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(INTPT),pt,INTORDER*sizeof(float),0,hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(INTWGT),wgt,INTORDER*sizeof(float),0,hipMemcpyHostToDevice));
    return EXIT_SUCCESS;
}

__host__ __device__ hipFloatComplex green(const float k, const cartCoord x, const cartCoord y)
{
    float r = cartDist(x,y);
    hipFloatComplex numerator = cplxExp(k*r);
    float denomenator = 4*PI*r;
    return make_hipFloatComplex(hipCrealf(numerator)/denomenator,hipCimagf(numerator)/denomenator);
}

__host__ hipFloatComplex triElemIntegral_g_nsgl(const float wavNum, const cartCoord nod[3], const cartCoord y, 
        const float *pt, const float *wgt)
{
    float J = cartNorm(crossProd(cartCoordSub(nod[0],nod[2]),cartCoordSub(nod[1],nod[2])));
    float rho, theta, eta1, eta2, xi1, xi2, xi3, wn, wm, temp;
    hipFloatComplex g, result = make_hipFloatComplex(0,0);
    cartCoord x;
    for(int n=0;n<INTORDER;n++) {
        eta2 = pt[n];
        wn = wgt[n];
        theta = 0.5f+0.5f*eta2;
        for(int m=0;m<INTORDER;m++) {
            eta1 = pt[m];
            wm = wgt[m];
            rho = 0.5f+0.5f*eta1;
            temp = 0.25f*wn*wm*rho*J;
            xi1 = rho*(1-theta);
            xi2 = rho-xi1;
            xi3 = 1-xi1-xi2;
            x = {
                    nod[0].x*xi1+nod[1].x*xi2+nod[2].x*xi3,
                    nod[0].y*xi1+nod[1].y*xi2+nod[2].y*xi3,
                    nod[0].z*xi1+nod[1].z*xi2+nod[2].z*xi3
                };
            g = green(wavNum,x,y);
            result = hipCaddf(result,make_hipFloatComplex(temp*hipCrealf(g),temp*hipCimagf(g)));
        }
    }
    return result;
}

__device__ hipFloatComplex triElemIntegral_g_nsgl(const float wavNum, const cartCoord nod[3], const cartCoord y)
{
    float J = cartNorm(crossProd(cartCoordSub(nod[0],nod[2]),cartCoordSub(nod[1],nod[2])));
    float rho, theta, eta1, eta2, xi1, xi2, xi3, wn, wm, temp;
    hipFloatComplex g, result = make_hipFloatComplex(0,0);
    cartCoord x;
    for(int n=0;n<INTORDER;n++) {
        eta2 = INTPT[n];
        wn = INTWGT[n];
        theta = 0.5f+0.5f*eta2;
        for(int m=0;m<INTORDER;m++) {
            eta1 = INTPT[m];
            wm = INTWGT[m];
            rho = 0.5f+0.5f*eta1;
            temp = 0.25f*wn*wm*rho*J;
            xi1 = rho*(1-theta);
            xi2 = rho-xi1;
            xi3 = 1-xi1-xi2;
            x = {
                    nod[0].x*xi1+nod[1].x*xi2+nod[2].x*xi3,
                    nod[0].y*xi1+nod[1].y*xi2+nod[2].y*xi3,
                    nod[0].z*xi1+nod[1].z*xi2+nod[2].z*xi3
                };
            g = green(wavNum,x,y);
            result = hipCaddf(result,make_hipFloatComplex(temp*hipCrealf(g),temp*hipCimagf(g)));
        }
    }
    return result;
}

__host__ hipFloatComplex triElemIntegral_g_sgl_3(const float wavNum, const cartCoord nod[3], 
        const float *pt, const float *wgt)
{
    float J = cartNorm(crossProd(cartCoordSub(nod[0],nod[2]),cartCoordSub(nod[1],nod[2])));
    float rho, theta, eta1, eta2, xi1, xi2, xi3, wn, wm, temp;
    hipFloatComplex g, result = make_hipFloatComplex(0,0);
    cartCoord x;
    for(int n=0;n<INTORDER;n++) {
        eta2 = pt[n];
        wn = wgt[n];
        theta = 0.5f+0.5f*eta2;
        for(int m=0;m<INTORDER;m++) {
            eta1 = pt[m];
            wm = wgt[m];
            rho = 0.5f+0.5f*eta1;
            temp = 0.25f*wn*wm*rho*J;
            xi1 = rho*(1-theta);
            xi2 = rho-xi1;
            xi3 = 1-xi1-xi2;
            x = {
                    nod[0].x*xi1+nod[1].x*xi2+nod[2].x*xi3,
                    nod[0].y*xi1+nod[1].y*xi2+nod[2].y*xi3,
                    nod[0].z*xi1+nod[1].z*xi2+nod[2].z*xi3
                };
            g = green(wavNum,x,nod[2]);
            result = hipCaddf(result,make_hipFloatComplex(temp*hipCrealf(g),temp*hipCimagf(g)));
        }
    }
    return result;
}

__device__ hipFloatComplex triElemIntegral_g_sgl_3(const float wavNum, const cartCoord nod[3])
{
    float J = cartNorm(crossProd(cartCoordSub(nod[0],nod[2]),cartCoordSub(nod[1],nod[2])));
    float rho, theta, eta1, eta2, xi1, xi2, xi3, wn, wm, temp;
    hipFloatComplex g, result = make_hipFloatComplex(0,0);
    cartCoord x;
    for(int n=0;n<INTORDER;n++) {
        eta2 = INTPT[n];
        wn = INTWGT[n];
        theta = 0.5f+0.5f*eta2;
        for(int m=0;m<INTORDER;m++) {
            eta1 = INTPT[m];
            wm = INTWGT[m];
            rho = 0.5f+0.5f*eta1;
            temp = 0.25f*wn*wm*rho*J;
            xi1 = rho*(1-theta);
            xi2 = rho-xi1;
            xi3 = 1-xi1-xi2;
            x = {
                    nod[0].x*xi1+nod[1].x*xi2+nod[2].x*xi3,
                    nod[0].y*xi1+nod[1].y*xi2+nod[2].y*xi3,
                    nod[0].z*xi1+nod[1].z*xi2+nod[2].z*xi3
                };
            g = green(wavNum,x,nod[2]);
            result = hipCaddf(result,make_hipFloatComplex(temp*hipCrealf(g),temp*hipCimagf(g)));
        }
    }
    return result;
}

__host__ hipFloatComplex triElemIntegral_g_sgl(const float wavNum, const cartCoord nod[3], 
        const float *pt, const float *wgt)
{
    cartCoord y;
    cartCoord ctr23 = scalarMul(0.5,cartCoordAdd(nod[1],nod[2]));
    y = cartCoordAdd(nod[0],scalarMul(2.0f/3.0f,cartCoordSub(ctr23,nod[0])));
    hipFloatComplex result1 = make_hipFloatComplex(0,0), result2 = make_hipFloatComplex(0,0), 
            result3 = make_hipFloatComplex(0,0), result = make_hipFloatComplex(0,0);
    cartCoord nod_sub[3];
    nod_sub[0] = nod[0];
    nod_sub[1] = nod[1];
    nod_sub[2] = y;
    result1 = triElemIntegral_g_sgl_3(wavNum,nod_sub,pt,wgt);
    nod_sub[0] = nod[1];
    nod_sub[1] = nod[2];
    nod_sub[2] = y;
    result2 = triElemIntegral_g_sgl_3(wavNum,nod_sub,pt,wgt);
    nod_sub[0] = nod[2];
    nod_sub[1] = nod[0];
    nod_sub[2] = y;
    result3 = triElemIntegral_g_sgl_3(wavNum,nod_sub,pt,wgt);
    result = hipCaddf(hipCaddf(result1,result2),result3);
    return result;
}

__device__ hipFloatComplex triElemIntegral_g_sgl(const float wavNum, const cartCoord nod[3])
{
    cartCoord y;
    cartCoord ctr23 = scalarMul(0.5,cartCoordAdd(nod[1],nod[2]));
    y = cartCoordAdd(nod[0],scalarMul(2.0f/3.0f,cartCoordSub(ctr23,nod[0])));
    hipFloatComplex result1 = make_hipFloatComplex(0,0), result2 = make_hipFloatComplex(0,0), 
            result3 = make_hipFloatComplex(0,0), result = make_hipFloatComplex(0,0);
    cartCoord nod_sub[3];
    nod_sub[0] = nod[0];
    nod_sub[1] = nod[1];
    nod_sub[2] = y;
    result1 = triElemIntegral_g_sgl_3(wavNum,nod_sub);
    nod_sub[0] = nod[1];
    nod_sub[1] = nod[2];
    nod_sub[2] = y;
    result2 = triElemIntegral_g_sgl_3(wavNum,nod_sub);
    nod_sub[0] = nod[2];
    nod_sub[1] = nod[0];
    nod_sub[2] = y;
    result3 = triElemIntegral_g_sgl_3(wavNum,nod_sub);
    result = hipCaddf(hipCaddf(result1,result2),result3);
    return result;
}

__host__ hipFloatComplex triElemIntegral_R(const float wavNum, const cartCoord nod[3], 
        const int n, const int m, const cartCoord x_lp, const float *pt, const float *wgt)
{
    float J = cartNorm(crossProd(cartCoordSub(nod[0],nod[2]),cartCoordSub(nod[1],nod[2])));
    float rho, theta, eta1, eta2, xi1, xi2, xi3, wn, wm, temp;
    hipFloatComplex rgl, result = make_hipFloatComplex(0,0);
    cartCoord x;
    sphCoord sphTemp;
    for(int i=0;i<INTORDER;i++) {
        eta2 = pt[i];
        wn = wgt[i];
        theta = 0.5f+0.5f*eta2;
        for(int j=0;j<INTORDER;j++) {
            eta1 = pt[j];
            wm = wgt[j];
            rho = 0.5f+0.5f*eta1;
            temp = 0.25f*wn*wm*rho*J;
            xi1 = rho*(1-theta);
            xi2 = rho-xi1;
            xi3 = 1-xi1-xi2;
            x = {
                    nod[0].x*xi1+nod[1].x*xi2+nod[2].x*xi3,
                    nod[0].y*xi1+nod[1].y*xi2+nod[2].y*xi3,
                    nod[0].z*xi1+nod[1].z*xi2+nod[2].z*xi3
                };
            sphTemp = cart2sph(cartCoordSub(x,x_lp));
            rgl = gsl_complex2cuFloatComplex(rglBasis(wavNum,n,m,sphTemp));
            result = hipCaddf(result,make_hipFloatComplex(temp*hipCrealf(rgl),temp*hipCimagf(rgl)));
        }
    }
    return result;
}

__host__ hipFloatComplex dirDev_R(const float wavNum, const int n, const int m, 
        const cartCoord nrml, const cartCoord x) 
{
    hipFloatComplex temp_c[6];
    float temp_f[6];
    sphCoord coord_sph = cart2sph(x);
    temp_c[0] = gsl_complex2cuFloatComplex(rglBasis(wavNum,n+1,m+1,coord_sph));
    if(n-1<0) {
        temp_c[1] = make_hipFloatComplex(0,0);
    } else {
        temp_c[1] = gsl_complex2cuFloatComplex(rglBasis(wavNum,n-1,m+1,coord_sph));
    }
    temp_c[2] = gsl_complex2cuFloatComplex(rglBasis(wavNum,n+1,m-1,coord_sph));
    if(n-1<0) {
        temp_c[3] = make_hipFloatComplex(0,0);
    } else {
        temp_c[3] = gsl_complex2cuFloatComplex(rglBasis(wavNum,n-1,m-1,coord_sph));
    }
    if(n-1<0) {
        temp_c[4] = make_hipFloatComplex(0,0);
    } else {
        temp_c[4] = gsl_complex2cuFloatComplex(rglBasis(wavNum,n-1,m,coord_sph));
    } 
    temp_c[5] = gsl_complex2cuFloatComplex(rglBasis(wavNum,n+1,m,coord_sph));
    temp_f[0] = bCoeff(n+1,-(m+1));
    temp_f[1] = bCoeff(n,m);
    temp_f[2] = bCoeff(n+1,m-1);
    temp_f[3] = bCoeff(n,-m);
    temp_f[4] = aCoeff(n-1,m);
    temp_f[5] = aCoeff(n,m);
    temp_c[0] = make_hipFloatComplex(hipCrealf(temp_c[0])*temp_f[0],hipCimagf(temp_c[0])*temp_f[0]);
    temp_c[1] = make_hipFloatComplex(hipCrealf(temp_c[1])*temp_f[1],hipCimagf(temp_c[1])*temp_f[1]);
    temp_c[2] = make_hipFloatComplex(hipCrealf(temp_c[2])*temp_f[2],hipCimagf(temp_c[2])*temp_f[2]);
    temp_c[3] = make_hipFloatComplex(hipCrealf(temp_c[3])*temp_f[3],hipCimagf(temp_c[3])*temp_f[3]);
    temp_c[4] = make_hipFloatComplex(hipCrealf(temp_c[4])*temp_f[4],hipCimagf(temp_c[4])*temp_f[4]);
    temp_c[5] = make_hipFloatComplex(hipCrealf(temp_c[5])*temp_f[5],hipCimagf(temp_c[5])*temp_f[5]);
    temp_c[0] = hipCsubf(temp_c[0],temp_c[1]);
    temp_c[1] = hipCsubf(temp_c[2],temp_c[3]);
    temp_c[2] = hipCsubf(temp_c[4],temp_c[5]);
    temp_c[3] = make_hipFloatComplex(wavNum/2.0f*nrml.x,-wavNum/2.0f*nrml.y);
    temp_c[4] = make_hipFloatComplex(wavNum/2.0f*nrml.x,wavNum/2.0f*nrml.y);
    temp_c[0] = hipCmulf(temp_c[3],temp_c[0]);
    temp_c[1] = hipCmulf(temp_c[4],temp_c[1]);
    temp_c[2] = make_hipFloatComplex(wavNum*nrml.z*hipCrealf(temp_c[2]),wavNum*nrml.z*hipCimagf(temp_c[2]));
    temp_c[0] = hipCaddf(temp_c[0],temp_c[1]);
    temp_c[0] = hipCaddf(temp_c[0],temp_c[2]);
    return temp_c[0];
}

__host__ hipFloatComplex triElemIntegral_pRpn(const float wavNum, const cartCoord nod[3], 
        const int n, const int m, const cartCoord x_lp, const float *pt, const float *wgt)
{
    cartCoord nrml = crossProd(cartCoordSub(nod[0],nod[2]),cartCoordSub(nod[1],nod[2]));
    cartCoord nrml_nrmlzd = normalize(nrml);
    float J = cartNorm(nrml);
    float rho, theta, eta1, eta2, xi1, xi2, xi3, wn, wm, temp;
    hipFloatComplex pRpn, result = make_hipFloatComplex(0,0);
    cartCoord x;
    for(int i=0;i<INTORDER;i++) {
        eta2 = pt[i];
        wn = wgt[i];
        theta = 0.5f+0.5f*eta2;
        for(int j=0;j<INTORDER;j++) {
            eta1 = pt[j];
            wm = wgt[j];
            rho = 0.5f+0.5f*eta1;
            temp = 0.25f*wn*wm*rho*J;
            xi1 = rho*(1-theta);
            xi2 = rho-xi1;
            xi3 = 1-xi1-xi2;
            x = {
                    nod[0].x*xi1+nod[1].x*xi2+nod[2].x*xi3,
                    nod[0].y*xi1+nod[1].y*xi2+nod[2].y*xi3,
                    nod[0].z*xi1+nod[1].z*xi2+nod[2].z*xi3
                };
            x = cartCoordSub(x,x_lp);
            pRpn = dirDev_R(wavNum,n,m,nrml_nrmlzd,x);
            result = hipCaddf(result,make_hipFloatComplex(temp*hipCrealf(pRpn),temp*hipCimagf(pRpn)));
        }
    }
    return result;
}

__host__ __device__ float prpn1(const cartCoord n, const cartCoord x, const cartCoord y)
{
    float dist = cartDist(x,y);
    return ((x.x-y.x)*n.x+(x.y-y.y)*n.y+(x.z-y.z)*n.z)/dist;
}

__host__ __device__ float prpn2(const cartCoord n, const cartCoord x, const cartCoord y)
{
    float dist = cartDist(x,y);
    return ((y.x-x.x)*n.x+(y.y-x.y)*n.y+(y.z-x.z)*n.z)/dist;
}

__host__ __device__ float prRecippn1(const cartCoord n, const cartCoord x, const cartCoord y)
{
    float dist = cartDist(x,y);
    return -1.0f/(dist*dist)*prpn1(n,x,y);
}

__host__ __device__ float prRecippn2(const cartCoord n, const cartCoord x, const cartCoord y)
{
    float dist = cartDist(x,y);
    return -1.0f/(dist*dist)*prpn2(n,x,y);
}

__host__ __device__ hipFloatComplex pGpn1(const float wavNum, const cartCoord x, const cartCoord y, 
        const cartCoord n)
{
    hipFloatComplex temp_c[2];
    float temp_f, dist = cartDist(x,y);
    temp_c[0] = green(wavNum,x,y);
    temp_c[1] = make_hipFloatComplex(-1.0f/dist,wavNum);
    temp_f = prpn1(n,x,y);
    temp_c[0] = hipCmulf(temp_c[0],temp_c[1]);
    temp_c[0] = make_hipFloatComplex(hipCrealf(temp_c[0])*temp_f,hipCimagf(temp_c[0])*temp_f);
    return temp_c[0];
}

__host__ __device__ hipFloatComplex pGpn2(const float wavNum, const cartCoord n, 
        const cartCoord x, const cartCoord y)
{
    hipFloatComplex temp_c[2];
    float temp_f, dist = cartDist(x,y);
    temp_c[0] = green(wavNum,x,y);
    temp_c[1] = make_hipFloatComplex(-1.0f/dist,wavNum);
    temp_f = prpn2(n,x,y);
    temp_c[0] = hipCmulf(temp_c[0],temp_c[1]);
    temp_c[0] = make_hipFloatComplex(hipCrealf(temp_c[0])*temp_f,hipCimagf(temp_c[0])*temp_f);
    return temp_c[0];
}

__host__ __device__ hipFloatComplex triElemIntegral_pGpn1_nsgl(const float wavNum, 
        const cartCoord nod[3], const cartCoord y, const float *pt, const float *wgt)
{
    cartCoord nrml = crossProd(cartCoordSub(nod[0],nod[2]),cartCoordSub(nod[1],nod[2]));
    cartCoord nrml_nrmlzd = normalize(nrml);
    float J = cartNorm(nrml);
    float rho, theta, eta1, eta2, xi1, xi2, xi3, wn, wm, temp;
    hipFloatComplex pgpn1, result = make_hipFloatComplex(0,0);
    cartCoord x;
    for(int n=0;n<INTORDER;n++) {
        eta2 = pt[n];
        wn = wgt[n];
        theta = 0.5f+0.5f*eta2;
        for(int m=0;m<INTORDER;m++) {
            eta1 = pt[m];
            wm = wgt[m];
            rho = 0.5f+0.5f*eta1;
            temp = 0.25f*wn*wm*rho*J;
            xi1 = rho*(1-theta);
            xi2 = rho-xi1;
            xi3 = 1-xi1-xi2;
            x = {
                    nod[0].x*xi1+nod[1].x*xi2+nod[2].x*xi3,
                    nod[0].y*xi1+nod[1].y*xi2+nod[2].y*xi3,
                    nod[0].z*xi1+nod[1].z*xi2+nod[2].z*xi3
                };
            pgpn1 = pGpn1(wavNum,x,y,nrml_nrmlzd);
            result = hipCaddf(result,make_hipFloatComplex(temp*hipCrealf(pgpn1),
                    temp*hipCimagf(pgpn1)));
        }
    }
    return result;
}

__host__ __device__ hipFloatComplex triElemIntegral_pGpn1_sgl_3(const float wavNum, const cartCoord nod[3], 
        const float *pt, const float *wgt)
{
    cartCoord nrml = crossProd(cartCoordSub(nod[0],nod[2]),cartCoordSub(nod[1],nod[2]));
    cartCoord nrml_nrmlzd = normalize(nrml);
    float J = cartNorm(nrml);
    float rho, theta, eta1, eta2, xi1, xi2, xi3, wn, wm, temp;
    hipFloatComplex pgpn1, result = make_hipFloatComplex(0,0);
    cartCoord x;
    for(int n=0;n<INTORDER;n++) {
        eta2 = pt[n];
        wn = wgt[n];
        theta = 0.5f+0.5f*eta2;
        for(int m=0;m<INTORDER;m++) {
            eta1 = pt[m];
            wm = wgt[m];
            rho = 0.5f+0.5f*eta1;
            temp = 0.25f*wn*wm*rho*J;
            xi1 = rho*(1-theta);
            xi2 = rho-xi1;
            xi3 = 1-xi1-xi2;
            x = {
                    nod[0].x*xi1+nod[1].x*xi2+nod[2].x*xi3,
                    nod[0].y*xi1+nod[1].y*xi2+nod[2].y*xi3,
                    nod[0].z*xi1+nod[1].z*xi2+nod[2].z*xi3
                };
            pgpn1 = pGpn1(wavNum,x,nod[2],nrml_nrmlzd);
            result = hipCaddf(result,make_hipFloatComplex(temp*hipCrealf(pgpn1),
                    temp*hipCimagf(pgpn1)));
        }
    }
    return result;
}

__host__ hipFloatComplex triElemIntegral_pgpn1_sgl(const float wavNum, const cartCoord nod[3], 
        const float *pt, const float *wgt)
{
    cartCoord y;
    cartCoord ctr23 = scalarMul(0.5,cartCoordAdd(nod[1],nod[2]));
    y = cartCoordAdd(nod[0],scalarMul(2.0f/3.0f,cartCoordSub(ctr23,nod[0])));
    hipFloatComplex result1 = make_hipFloatComplex(0,0), result2 = make_hipFloatComplex(0,0), 
            result3 = make_hipFloatComplex(0,0), result = make_hipFloatComplex(0,0);
    cartCoord nod_sub[3];
    nod_sub[0] = nod[0];
    nod_sub[1] = nod[1];
    nod_sub[2] = y;
    result1 = triElemIntegral_pGpn1_sgl_3(wavNum,nod_sub,pt,wgt);
    nod_sub[0] = nod[1];
    nod_sub[1] = nod[2];
    nod_sub[2] = y;
    result2 = triElemIntegral_pGpn1_sgl_3(wavNum,nod_sub,pt,wgt);
    nod_sub[0] = nod[2];
    nod_sub[1] = nod[0];
    nod_sub[2] = y;
    result3 = triElemIntegral_pGpn1_sgl_3(wavNum,nod_sub,pt,wgt);
    result = hipCaddf(hipCaddf(result1,result2),result3);
    return result;
}

__host__ __device__ hipFloatComplex triElemIntegral_pGpn2_nsgl(const float wavNum, 
        const cartCoord nod[3], const cartCoord nrml, const cartCoord y, 
        const float *pt, const float *wgt)
{
    float J = cartNorm(crossProd(cartCoordSub(nod[0],nod[2]),cartCoordSub(nod[1],nod[2])));
    float rho, theta, eta1, eta2, xi1, xi2, xi3, wn, wm, temp;
    hipFloatComplex pgpn2, result = make_hipFloatComplex(0,0);
    cartCoord x;
    for(int n=0;n<INTORDER;n++) {
        eta2 = pt[n];
        wn = wgt[n];
        theta = 0.5f+0.5f*eta2;
        for(int m=0;m<INTORDER;m++) {
            eta1 = pt[m];
            wm = wgt[m];
            rho = 0.5f+0.5f*eta1;
            temp = 0.25f*wn*wm*rho*J;
            xi1 = rho*(1-theta);
            xi2 = rho-xi1;
            xi3 = 1-xi1-xi2;
            x = {
                    nod[0].x*xi1+nod[1].x*xi2+nod[2].x*xi3,
                    nod[0].y*xi1+nod[1].y*xi2+nod[2].y*xi3,
                    nod[0].z*xi1+nod[1].z*xi2+nod[2].z*xi3
                };
            pgpn2 = pGpn2(wavNum,nrml,x,y);
            result = hipCaddf(result,make_hipFloatComplex(temp*hipCrealf(pgpn2),
                    temp*hipCimagf(pgpn2)));
        }
    }
    return result;
}

__host__ __device__ hipFloatComplex triElemIntegral_pGpn2_sgl_3(const float wavNum, 
        const cartCoord nod[3], const float *pt, const float *wgt)
{
    cartCoord nrml = crossProd(cartCoordSub(nod[0],nod[2]),cartCoordSub(nod[1],nod[2]));
    cartCoord nrml_nrmlzd = normalize(nrml);
    float J = cartNorm(nrml);
    float rho, theta, eta1, eta2, xi1, xi2, xi3, wn, wm, temp;
    hipFloatComplex pgpn2, result = make_hipFloatComplex(0,0);
    cartCoord x;
    for(int n=0;n<INTORDER;n++) {
        eta2 = pt[n];
        wn = wgt[n];
        theta = 0.5f+0.5f*eta2;
        for(int m=0;m<INTORDER;m++) {
            eta1 = pt[m];
            wm = wgt[m];
            rho = 0.5f+0.5f*eta1;
            temp = 0.25f*wn*wm*rho*J;
            xi1 = rho*(1-theta);
            xi2 = rho-xi1;
            xi3 = 1-xi1-xi2;
            x = {
                    nod[0].x*xi1+nod[1].x*xi2+nod[2].x*xi3,
                    nod[0].y*xi1+nod[1].y*xi2+nod[2].y*xi3,
                    nod[0].z*xi1+nod[1].z*xi2+nod[2].z*xi3
                };
            pgpn2 = pGpn2(wavNum,nrml_nrmlzd,x,nod[2]);
            result = hipCaddf(result,make_hipFloatComplex(temp*hipCrealf(pgpn2),
                    temp*hipCimagf(pgpn2)));
        }
    }
    return result;
}

__host__ hipFloatComplex triElemIntegral_pgpn2_sgl(const float wavNum, 
        const cartCoord nod[3], const float *pt, const float *wgt)
{
    cartCoord y;
    cartCoord ctr23 = scalarMul(0.5,cartCoordAdd(nod[1],nod[2]));
    y = cartCoordAdd(nod[0],scalarMul(2.0f/3.0f,cartCoordSub(ctr23,nod[0])));
    hipFloatComplex result1 = make_hipFloatComplex(0,0), result2 = make_hipFloatComplex(0,0), 
            result3 = make_hipFloatComplex(0,0), result = make_hipFloatComplex(0,0);
    cartCoord nod_sub[3];
    nod_sub[0] = nod[0];
    nod_sub[1] = nod[1];
    nod_sub[2] = y;
    result1 = triElemIntegral_pGpn2_sgl_3(wavNum,nod_sub,pt,wgt);
    nod_sub[0] = nod[1];
    nod_sub[1] = nod[2];
    nod_sub[2] = y;
    result2 = triElemIntegral_pGpn2_sgl_3(wavNum,nod_sub,pt,wgt);
    nod_sub[0] = nod[2];
    nod_sub[1] = nod[0];
    nod_sub[2] = y;
    result3 = triElemIntegral_pGpn2_sgl_3(wavNum,nod_sub,pt,wgt);
    result = hipCaddf(hipCaddf(result1,result2),result3);
    return result;
}

__host__ __device__ hipFloatComplex p2Gpn1pn2(const float wavNum, const cartCoord n1, const cartCoord n2, 
        const cartCoord x, const cartCoord y)
{
    hipFloatComplex temp[3];
    float dist, t;
    
    dist = cartDist(x,y);
    temp[0] = cplxExp(wavNum*dist);
    t = 4*PI*dist*dist*dist;
    temp[0] = make_hipFloatComplex(hipCrealf(temp[0])/t,hipCimagf(temp[0])/t);
    
    temp[1] = make_hipFloatComplex(3-wavNum*wavNum*dist*dist,-3*wavNum*dist);
    t = prpn1(n1,x,y)*prpn2(n2,x,y);
    temp[1] = make_hipFloatComplex(hipCrealf(temp[1])*t,hipCimagf(temp[1])*t);
    
    t = dotProd(n1,n2);
    temp[2] = make_hipFloatComplex(1,-wavNum*dist);
    temp[2] = make_hipFloatComplex(hipCrealf(temp[2])*t,hipCimagf(temp[2])*t);
    
    return hipCmulf(temp[0],hipCaddf(temp[1],temp[2]));
}

__host__ __device__ hipFloatComplex triElemIntegral_p2Gpn1pn2_nsgl(const float wavNum, 
        const cartCoord nod[3], const cartCoord nrml_y, const cartCoord y, 
        const float *pt, const float *wgt)
{
    cartCoord nrml_x = crossProd(cartCoordSub(nod[0],nod[2]),cartCoordSub(nod[1],nod[2]));
    cartCoord nrml_x_nrmlzd = normalize(nrml_x);
    float J = cartNorm(nrml_x);
    float rho, theta, eta1, eta2, xi1, xi2, xi3, wn, wm, temp;
    hipFloatComplex p2gpn1pn2, result = make_hipFloatComplex(0,0);
    cartCoord x;
    for(int n=0;n<INTORDER;n++) {
        eta2 = pt[n];
        wn = wgt[n];
        theta = 0.5f+0.5f*eta2;
        for(int m=0;m<INTORDER;m++) {
            eta1 = pt[m];
            wm = wgt[m];
            rho = 0.5f+0.5f*eta1;
            temp = 0.25f*wn*wm*rho*J;
            xi1 = rho*(1-theta);
            xi2 = rho-xi1;
            xi3 = 1-xi1-xi2;
            x = {
                    nod[0].x*xi1+nod[1].x*xi2+nod[2].x*xi3,
                    nod[0].y*xi1+nod[1].y*xi2+nod[2].y*xi3,
                    nod[0].z*xi1+nod[1].z*xi2+nod[2].z*xi3
                };
            p2gpn1pn2 = p2Gpn1pn2(wavNum,nrml_x_nrmlzd,nrml_y,x,y);
            result = hipCaddf(result,make_hipFloatComplex(temp*hipCrealf(p2gpn1pn2),
                    temp*hipCimagf(p2gpn1pn2)));
        }
    }
    return result;
}






